#include "hip/hip_runtime.h"
#include "definition/mmpool_size.h"
#include "generation/gen_label.cuh"
#include <hip/hip_runtime.h>

#define cuda_block_dim 256

__global__ void
gen_labels_kernel_vertex_level(gpu_Graph *d_g, cuda_vector<hub_type> *L_gpu,
                               cuda_hashTable<int, int> *L_hashes,
                               cuda_queue<hub_type> *Qs, int upper_bound) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= d_g->num_nodes)
    return;
  // printf("Thread %d starting\n", tid);
  cuda_queue<hub_type> *Q1 = Qs + tid;
  cuda_hashTable<int, int> *L_hash = L_hashes + tid;

  Q1->enqueue({tid, 0, 0, 0});
  L_gpu[tid].push_back({tid, tid, 0, 0});

  int degree_tid = d_g->d_offsets[tid + 1] - d_g->d_offsets[tid];
  while (!Q1->is_empty()) {
    hub_type temp;
    Q1->dequeue(&temp);
    int u = temp.hub_vertex;
    if (L_hash->find(u) != -1)
      continue;
    L_hash->insert(u, 1);
    weight_type du = temp.distance;
    int hu = temp.hop;

    int degree_u = d_g->d_offsets[u + 1] - d_g->d_offsets[u];

    if (degree_tid >= degree_u) {
      hub_type t1, t2;
      weight_type q_dis;
      if (tid == u) {
        q_dis = 0.0;
      } else {
        query_mindis_with_hub_device(upper_bound, L_gpu+tid, L_gpu+u, &t1,
                                     &t2, &q_dis);
      }
      if (q_dis > du) {
        // 插入标签到L_gpu
        L_gpu[tid].push_back({u, tid, hu, du});
      }

      int h1 = hu + 1;
      if (h1 <= upper_bound) {
        for (int i = d_g->d_offsets[u]; i < d_g->d_offsets[u + 1]; ++i) {
          int v = d_g->d_edges[i].target;
          weight_type dv = du + d_g->d_edges[i].weight;
          hub_type t1, t2;
          weight_type q_dis_v;

          if (tid == v) {
            q_dis_v = 0.0;
          } else {
            query_mindis_with_hub_device(upper_bound, L_gpu+tid, L_gpu+v, &t1,
                                         &t2, &q_dis_v);
          }
          if (q_dis_v > dv && L_hash->find(v) == -1) {
            Q1->enqueue({v, u, h1, dv});
          }
        }
      }
    }
  }
}

void gen_labels_gpu(graph_v_of_v<weight_type> *G,
                    hop_constrained_case_info *info, int upper_bound) {

  int vertex_num = G->size();
  //int edge_num = G->edge_number();

  //生成gpu上的graph
  // 将邻接表转换为一维数组表示的图结构

  gpu_Graph *d_g;
  hipMallocManaged(&d_g, sizeof(gpu_Graph));
  new (d_g) gpu_Graph(G->ADJs);

  // init case_info
  info = new hop_constrained_case_info();
  info->init(vertex_num, d_g->max_degree * vertex_num / nodes_per_block +
                             vertex_num + 3000);

  printf("init case_info success\n");

  //准备queues
  cuda_queue<hub_type> *queues;
  hipMallocManaged(&queues, vertex_num * sizeof(cuda_queue<hub_type>));

  int queue_size_blocks = d_g->max_degree / nodes_per_block + 1;
  printf("queue_size: %d\n", queue_size_blocks);

  for (int i = 0; i < vertex_num; i++) {

    new (&queues[i])
        cuda_queue<hub_type>(queue_size_blocks, info->mmpool_labels);
  }

  //准备hashTable
  printf("max_degree: %d\n", d_g->max_degree);
  cuda_hashTable<int, int> *L_hash;
  hipMallocManaged(&L_hash, vertex_num * sizeof(cuda_hashTable<int, int>));
  for (int i = 0; i < vertex_num; i++) {
    new (L_hash + i) cuda_hashTable<int, int>(d_g->max_degree * 10);
  }
  // printf L_hash[0].table[0].first

  // 确保初始化完成
  hipDeviceSynchronize();

  printf("initation success\n");
  int grid_dim = (vertex_num + cuda_block_dim - 1) / cuda_block_dim;
  gen_labels_kernel_vertex_level<<<grid_dim, cuda_block_dim>>>(
      d_g, info->L_cuda, L_hash, queues, upper_bound);

  hipError_t err = hipGetLastError(); // 检查内核启动错误
  if (err != hipSuccess) {
    std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err)
              << std::endl;
  }

  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(err));
    // 在这里添加更多调试信息
  }

  printf("task success\n");
  //拷贝结果到cpu
  // info->vector_gpu_to_cpu();
  printf("copy success\n");

  // //打印
  printf("dis , hop , hub , parent \n");
  for (int i = 0; i < vertex_num; i++) {
    printf("vertex %d\n", i);
    for (int j = 0; j < info->L_cuda[i].size(); j++) {
      printf("{%d, %d, %d,%d},",
             (info->L_cuda[i]).get(j)->distance, (info->L_cuda[i]).get(j)->hop,
             (info->L_cuda[i]).get(j)->hub_vertex,
             (info->L_cuda[i]).get(j)->parent_vertex);
    }
    printf("\n");
  }

  //释放内存
  info->destroy_L_cuda();
  for(int i = 0; i < vertex_num; i++){
    queues[i].~cuda_queue();
    L_hash[i].~cuda_hashTable();
  }
  hipFree(L_hash);
  hipFree(queues);
  hipFree(d_g);
  free(info);
  

  return;
}