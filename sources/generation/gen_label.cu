#include "hip/hip_runtime.h"
#include "definition/hub_def.h"
#include "definition/mmpool_size.h"
#include "generation/cuda_clean_label.cuh"
#include "generation/gen_label.cuh"
#include "graph/graph_v_of_v.h"
#include "memoryManagement/cuda_hashTable.cuh"
#include "utilities/dijkstra.h"
#include <cassert>
#include <hip/hip_runtime.h>

#define cuda_block_dim 256

void correctness_check(hop_constrained_case_info *info,
                       graph_v_of_v<disType> *G, int upper_bound) {
  dijkstra_table d_t(*G, false, upper_bound);
  //随机选择100个起点，100个终点，计算最短路径
  vector<int> start_points;
  vector<int> end_points;

  for (int i = 0; i < 100; i++) {
    start_points.push_back(rand() % G->size());
    end_points.push_back(rand() % G->size());
  }

  for (int i = 0; i < 100; i++) {
    int start = start_points[i];
    int end = end_points[i];
    // printf("start: %d, end: %d\n", start, end);
    //计算最短路径
    d_t.runDijkstra(start);
    disType dis1 = d_t.query_distance(start, end);
    disType dis2 = info->query_distance(start, end, upper_bound);
    if (abs(dis1 - dis2) > 1e-6) {
      info->print_final_label(start);
      info->print_final_label(end);
      vector<int> path = d_t.query_path(start, end);
      printf("path: ");
      for (int i = 0; i < path.size(); i++) {
        printf("%d ", path[i]);
      }
      printf("start: %d, end: %d\n", start, end);
      printf("dis1: %d, dis2: %d\n", dis1, dis2);
      assert(0);
    }
  }
}

__global__ void gen_labels_kernel_vertex_level(
    gpu_Graph *d_g, cuda_vector<hub_type> *L_gpu,
    cuda_hashTable<int, int> *L_hashes, cuda_queue<hub_type> *Qs,
    int upper_bound, int *group, cuda_hashTable<int, int> *reflect_group,
    int vertex_num) {

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= d_g->num_nodes || tid >= vertex_num)
    return;

  int g_id = group[tid];
  // printf("Thread %d starting\n", tid);
  cuda_queue<hub_type> *Q1 = Qs + tid;
  cuda_hashTable<int, int> *L_hash = L_hashes + tid;

  Q1->enqueue({tid, tid, 0, 0}); // group id
  L_gpu[tid].push_back({g_id, g_id, 0, 0});

  int degree_v_id = d_g->d_offsets[g_id + 1] - d_g->d_offsets[g_id];

  while (!Q1->is_empty()) {
    hub_type temp;
    Q1->dequeue(&temp);
    int u = temp.hub_vertex; //代表在graph中的id
    int g_u = group[u];

    if (L_hash->find(u) != NULL)
      continue;
    L_hash->insert(u, 1);
    weight_type du = temp.distance;
    int hu = temp.hop;

    int degree_u = d_g->d_offsets[g_u + 1] -
                   d_g->d_offsets[g_u]; //与图交互，需要使用图中的id

    if (degree_v_id > degree_u || (degree_v_id == degree_u && g_id >= g_u)) {
      hub_type t1, t2;
      weight_type q_dis;
      if (tid == u) {
        q_dis = 0.0;
      } else {
        query_mindis_with_hub_device(upper_bound, L_gpu + tid, L_gpu + u, &t1,
                                     &t2, &q_dis);
      }
      if (q_dis > du) {
        // 插入标签到L_gpu
        L_gpu[u].push_back(
            {g_id, temp.parent_vertex, hu, du}); //插入时，hub_vertex是图中
      }

      int h1 = hu + 1;
      if (h1 <= upper_bound) {
        for (int i = d_g->d_offsets[g_u]; i < d_g->d_offsets[g_u + 1]; ++i) {
          int v = d_g->d_edges[i].target;          //图中的id
          int *reflect_v = reflect_group->find(v); //转换为group中的id
          if (reflect_v == NULL) { //说明超过了group中的点，直接去除
            continue;
          }

          weight_type dv = du + d_g->d_edges[i].weight;
          hub_type t1, t2;
          weight_type q_dis_v;

          if (tid == v) {
            q_dis_v = 0.0;
          } else {
            query_mindis_with_hub_device(upper_bound, L_gpu + tid,
                                         L_gpu + *reflect_v, &t1, &t2,
                                         &q_dis_v);
          }
          if (q_dis_v > dv && L_hash->find(*reflect_v) == NULL) {
            Q1->enqueue({*reflect_v, g_u, h1, dv});
          }
        }
      }
    }
  }
}

void gen_labels_gpu(graph_v_of_v<weight_type> *G,
                    hop_constrained_case_info *info, int upper_bound) {

  hipError_t err;
  // 1. Initiation
  // 包括初始化gpu上的图结构，初始化case_info，初始化queues和hashTable

  int vertex_num = info->L_size;

  // int edge_num = G->edge_number();

  //生成gpu上的graph
  // 将邻接表转换为一维数组表示的图结构

  gpu_Graph *d_g;
  hipMallocManaged(&d_g, sizeof(gpu_Graph));
  new (d_g) gpu_Graph(G->ADJs);

  // printf("init case_info success\n");

  //准备queues
  cuda_queue<hub_type> *queues;
  hipMallocManaged(&queues, vertex_num * sizeof(cuda_queue<hub_type>));

  int queue_size_blocks = 100;
  // printf("queue_size: %d\n", queue_size_blocks);

  for (int i = 0; i < vertex_num; i++) {

    new (&queues[i])
        cuda_queue<hub_type>(queue_size_blocks, info->mmpool_labels);
  }

  //准备hashTable
  // printf("max_degree: %d\n", d_g->max_degree);
  cuda_hashTable<int, int> *L_hash;
  hipMallocManaged(&L_hash, vertex_num * sizeof(cuda_hashTable<int, int>));
  for (int i = 0; i < vertex_num; i++) {
    new (L_hash + i) cuda_hashTable<int, int>(d_g->max_degree * 10);
  }

  // 1. Initiation success
  // printf("initation success\n");

  // 2. Task，生成label

  int grid_dim = (vertex_num + cuda_block_dim - 1) / cuda_block_dim;
  // 时间测量
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  gen_labels_kernel_vertex_level<<<grid_dim, cuda_block_dim>>>(
      d_g, info->L_cuda, L_hash, queues, upper_bound, info->group,
      info->reflect_group, vertex_num);

  err = hipGetLastError(); // 检查内核启动错误
  if (err != hipSuccess) {
    std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err)
              << std::endl;
  }

  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(err));
    // 在这里添加更多调试信息
  }
  // 时间测量结束
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("generate label time: %f ms\n", milliseconds);

  //拷贝结果到cpu

  // 2. 生成label结束

  // 3. sort label,对标签进行清洗
  // 时间测量
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  //   hipEventRecord(start, 0);

  //   cuda_sort(info->L_cuda, vertex_num);
  //   err = hipGetLastError(); // 检查内核启动错误
  //   if (err != hipSuccess) {
  //     std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err)
  //               << std::endl;
  //   }

  //   err = hipDeviceSynchronize();
  //   if (err != hipSuccess) {
  //     std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err)
  //               << std::endl;
  //   }

  //   // 时间测量结束
  //   hipEventRecord(stop, 0);
  //   hipEventSynchronize(stop);
  //   hipEventElapsedTime(&milliseconds, start, stop);

  //   printf("sort label time: %f ms\n", milliseconds);

  //   // 3. sort label结束

  //   // // 4. clean label
  //   // clean之前，我们要先生成最后的cuda_label

  //   cuda_label<hub_type> *Labels;
  //   hipMallocManaged(&Labels, vertex_num * sizeof(cuda_label<hub_type>));
  //   for (int i = 0; i < vertex_num; i++) {
  //     new (Labels + i) cuda_label<hub_type>(info->L_cuda[i].first_elements,
  //                                           info->L_cuda[i].size(), i);
  //   }

  // //  时间测量
  //   hipEventCreate(&start);
  //   hipEventCreate(&stop);
  //   hipEventRecord(start, 0);

  //   cuda_clean_label<<<grid_dim, cuda_block_dim>>>(Labels, upper_bound,
  //   vertex_num);

  //   err = hipDeviceSynchronize();
  //   if (err != hipSuccess) {
  //     std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err)
  //               << std::endl;
  //   }

  //   // 时间测量结束
  //   hipEventRecord(stop, 0);
  //   hipEventSynchronize(stop);
  //   hipEventElapsedTime(&milliseconds, start, stop);

  //   for (int i = 0; i < vertex_num; i++) {
  //     Labels[i].minimize();
  //   }

  //   printf("clean label time: %f ms\n", milliseconds);

  //   // 4. clean label结束

  //   //  打印

  //   correctness_check(Labels, G, upper_bound);
  //   printf("hub ,dis , hop ,  parent \n");
  //   for (int i = 0; i < vertex_num; i++) {
  //     printf("label %d\n", i);
  //     Labels[i].print_L();
  //     Labels[i].print_index_table();
  //     printf("\n");
  //   }

  // clean
}