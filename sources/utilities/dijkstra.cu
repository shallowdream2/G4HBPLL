#include "hip/hip_runtime.h"
#include "utilities/dijkstra.cuh"
#include "definition/hub_def.h"
#include "utilities/GPU_csr.hpp"
#include <unordered_map>
#include <unordered_set>

dijkstra_table::dijkstra_table(graph_v_of_v<disType> &g, bool is_directed,
                               int k, vector<int> sources)
    : graph(g), is_directed(is_directed), k(k),input_graph(graph_v_of_v_to_CSR<disType>(g)),
      source_set(unordered_set<int>(sources.begin(), sources.end())) {
  
  for (int source : sources) {
    runDijkstra(source);
  }
}

void dijkstra_table::runDijkstra_gpu(vector<int> &sources) {
  for (int source : sources) {
    if (source_set.find(source) == source_set.end()) {
      source_set.insert(source);
      query_table_gpu[source].resize(graph.size());
      gpu_shortest_paths(input_graph, source, query_table_gpu[source]);
    }
  }
}

void dijkstra_table::runDijkstra(int s) {
  if (query_table_cpu.find(s) != query_table_cpu.end()) {
    return;
  }
  priority_queue<pair<disType, pair<int, int>>, // <dis, <当前节点, hop>>
                 vector<pair<disType, pair<int, int>>>,
                 greater<pair<disType, pair<int, int>>>>
      pq;
  unordered_map<int, entry> distance;

  // 初始化所有节点的距离为无穷大，前驱为-1
  for (int i = 0; i < graph.size(); i++) {
    distance[i] = {numeric_limits<disType>::max(), -1};
  }

  // 将源节点加入优先队列
  pq.push({0, {s, 0}});
  distance[s] = {0, s};

  while (!pq.empty()) {

    int u = pq.top().second.first;
    disType dist = pq.top().first;
    int hop = pq.top().second.second;
    pq.pop();
    if (dist > distance[u].first) {
      //说明distance已经被更新，可以直接跳出
      continue;
    }
    if (hop + 1 > this->k)
      continue; // 跳数限制
    // 遍历邻接节点
    for (auto &[v, weight] : graph[u]) {
      disType new_dist = dist + weight;
      if (new_dist < distance[v].first) {
        distance[v].first = new_dist;
        distance[v].second = u;
        pq.push({new_dist, {v, hop + 1}});
      }
    }
  }

  // 保存计算结果到查询表
  query_table_cpu[s] = distance;
}

vector<int> dijkstra_table::query_path(int s, int t) {
  vector<int> path;
  if (query_table_cpu.find(s) != query_table_cpu.end()) {
    int cur = t;
    while (cur != s) {
      path.push_back(cur);
      cur = query_table_cpu[s][cur].second;
    }
    path.push_back(s);
    reverse(path.begin(), path.end());

  } else if (query_table_cpu.find(t) != query_table_cpu.end()) {
    int cur = s;
    while (cur != t) {
      path.push_back(cur);
      cur = query_table_cpu[t][cur].second;
    }
    path.push_back(t);
  }
  return path;
}

disType dijkstra_table::query_distance(int s, int t) {
  if (is_gpu) {
    if (source_set.find(s) != source_set.end()) {
      return query_table_gpu[s][t];
    }
    if (source_set.find(t) != source_set.end()) {
      return query_table_gpu[t][s];
    }
    return numeric_limits<disType>::max();
  }
  disType min_distance = numeric_limits<disType>::max();

  if (query_table_cpu.find(s) != query_table_cpu.end()) {
    if (query_table_cpu[s].find(t) != query_table_cpu[s].end()) {
      min_distance = query_table_cpu[s][t].first;
    }
  } else if (query_table_cpu.find(t) != query_table_cpu.end()) {
    if (query_table_cpu[t].find(s) != query_table_cpu[t].end()) {
      min_distance = query_table_cpu[t][s].first;
    }
  }

  return min_distance;
}
