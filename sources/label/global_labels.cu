#include "definition/mmpool_size.h"
#include "label/global_labels.cuh"
#include "memoryManagement/cuda_vector.cuh"

__host__ void hop_constrained_case_info::init(int vertex_nums,int mmpool_size_block) {
  L_size = vertex_nums;

  hipMallocManaged(&mmpool_labels,
                    sizeof(mmpool<hub_type>)); // 创建统一内存的对象
  new (mmpool_labels) mmpool<hub_type>(mmpool_size_block); // 调用构造函数

  hipMallocManaged(
      &L_cuda, vertex_nums * sizeof(cuda_vector<hub_type> )); // 分配n个cuda_vector指针
  for (int i = 0; i < vertex_nums; i++) {

    new (L_cuda+i) cuda_vector<hub_type>(mmpool_labels,vertex_nums/nodes_per_block+1); // 调用构造函数
  }
   hipDeviceSynchronize();
}

__host__ void hop_constrained_case_info::destroy_L_cuda() {
  for (int i = 0; i < L_size; i++) {
    L_cuda[i].~cuda_vector<hub_type>(); // 调用析构函数
  }
  hipFree(L_cuda);
}



// __host__ void hop_constrained_case_info::vector_gpu_to_cpu() {
//   //将gpu的vector转移到cpu，先从gpu拷贝结果回来，然后再转移到cpu
//   L_cpu.clear();
//   L_cpu.resize(L_size, vector<hub_type>());
//   for (int i = 0; i < L_size; i++) {
//     for (int j = 0; j < L_cuda[i]->size(); j++) {
//       hub_type tmp;
//       L_cuda[i]->copy_to_cpu(j, &tmp);
//       // printf("tmp: %lf %d %d %d\n", tmp.distance, tmp.hop, tmp.hub_vertex,
//       //        tmp.parent_vertex);
//       L_cpu[i].push_back(tmp);
//     }
//   }
// }