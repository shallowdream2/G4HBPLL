#include "definition/hub_def.h"
#include "generation/gen_label.cuh"
#include <graph/graph_v_of_v.h>
#include <memoryManagement/cuda_label.cuh>
#define DATASET_PATH "/mnt/f/linux/rucgraph-HBPLL-GPU/data/euroroad2.txt"

int main() {

  // int deviceCount = 0;
  // hipError_t err = hipGetDeviceCount(&deviceCount);

  // if (err != hipSuccess) {
  //   printf("CUDA error: %s\n", hipGetErrorString(err));
  //   return -1;
  // }

  // printf("Detected %d CUDA Capable device(s)\n", deviceCount);

  // for (int i = 0; i < deviceCount; i++) {
  //   hipDeviceProp_t deviceProp;
  //   hipGetDeviceProperties(&deviceProp, i);
  //   printf("Device %d: %s\n", i, deviceProp.name);
  //   // 在这里可以打印出更多的设备属性
  // }
  // hipStream_t stream;
  // hipStreamCreate(&stream);
  // int *test1;
  // hipMallocManaged(&test1, sizeof(int));
  // *test1 = 1;
  // int *test2;
  // hipMallocManaged(&test2, sizeof(int));
  // size_t i = 0;
  // for (; i < 1;) {
  //   int start_idx = i;
  //   int end_idx = i;
  //   while (end_idx + 1 < 1 && 1 == 1) {
  //     end_idx++;
  //   }
  //   size_t continuous_block_size = (end_idx - start_idx + 1) * 1;
  //   if (end_idx == 1 - 1) {
  //     continuous_block_size = (end_idx - start_idx) * 1 + 1 % 1;
  //   }
  //   int *block_start_ptr = test1;
  //   hipError_t err = hipMemcpyAsync(test2, block_start_ptr,
  //                                     continuous_block_size * sizeof(int),
  //                                     hipMemcpyDefault, stream);
  //   i = end_idx + 1;
  // }

  // hipStreamSynchronize(stream);

  // printf("%d\n", *test2);

  graph_v_of_v<weight_type> instance_graph;
  instance_graph.txt_read(DATASET_PATH);
  printf("Graph read from %s\n", DATASET_PATH);
  printf("Number of vertices: %d\n", instance_graph.size());

  hop_constrained_case_info *info = NULL;
  //cuda_label<hub_type> *Labels = NULL;
  gen_labels_gpu(&instance_graph, info,  5);

  // printf
  //  //打印
  // printf("dis , hop , hub , parent \n");
  // for (int i = 0; i < vertex_num; i++) {
  //   info->L_cuda[i].sort_label();
  //   printf("vertex %d\n", i);
  //   for (int j = 0; j < info->L_cuda[i].size(); j++) {
  //     printf("{%d, %d, %d,%d},", (info->L_cuda[i]).get(j)->distance,
  //            (info->L_cuda[i]).get(j)->hop,
  //            (info->L_cuda[i]).get(j)->hub_vertex,
  //            (info->L_cuda[i]).get(j)->parent_vertex);
  //   }
  //   printf("\n");
  // }
}