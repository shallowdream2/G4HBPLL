#include "hip/hip_runtime.h"
#include "memoryManagement/cuda_vector.cuh" // 确保包含正确的头文件路径
#include "memoryManagement/mmpool.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

// 核函数，用于测试 cuda_vector 功能
__global__ void test_vector(cuda_vector<int> *vec) {
//  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  // 在向量中添加一些元素
  printf("vec->size() = %d\n", vec->size());
  vec->push_back(2);
  if (vec->push_back(1)) {
    printf("push_back(1) success\n");
  }
  printf("vec->size() = %d\n", vec->size());
  printf("add:%d\n",vec->operator[](0)+vec->operator[](1));


  // 确保所有线程都已完成写操作
  //__syncthreads();

  // if (idx < vec->size()) {
  //     printf("vec[%d] = %d\n", idx, (*vec)[idx]);
  // }
}

__global__ void test_pool(mmpool<int> *pool) {
//  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  // 在向量中添加一些元素
  printf("pool->size() = %d\n", pool->size());
  if (pool->push_node(2, 1)) {
    printf("push_node(2, 1) success\n");
  }

  // 确保所有线程都已完成写操作
  //__syncthreads();

  // if (idx < vec->size()) {
  //     printf("vec[%d] = %d\n", idx, (*vec)[idx]);
  // }
}

int main() {
  mmpool<int> *pool;
  hipMallocManaged(&pool, sizeof(mmpool<int>)); // 创建统一内存的对象
  new (pool) mmpool<int>(10, 100); // 调用构造函数

  test_pool<<<1,1>>>(pool);
  hipDeviceSynchronize(); // 等待核函数完成

  printf("%u\n",pool->size());

  
  // // 分配和初始化 cuda_vector
  cuda_vector<int> *d_vector;
  hipMallocManaged(&d_vector, sizeof(cuda_vector<int>));
  new (d_vector) cuda_vector<int>(pool); // 调用构造函数

  // 启动核函数
  test_vector<<<1, 1>>>(d_vector);

  // 等待 GPU 完成
  hipError_t error = hipDeviceSynchronize();
  if (error != hipSuccess) {
    std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
    return 1;
  }
  // 销毁 vector 和释放资源

  hipFree(d_vector);
  // 清理
  pool->~mmpool<int>(); // 调用析构函数
  hipFree(pool); // 释放内存


  return 0;
}
